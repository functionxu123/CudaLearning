#define CUDA_CALL(x)                                                         \
    {                                                                        \
        const hipError_t a = (x);                                           \
        if (a != hipSuccess) {                                              \
            printf("\nCUDA ERROR: %s (err_num=%d)\n", hipGetErrorString(a), \
                   a);                                                       \
            hipDeviceReset();                                               \
            assert(0);                                                       \
        }                                                                    \
    }

#include <stdio.h>

#include "hip/hip_runtime.h"


int main(){
    CUDA_CALL(hipSetDevice(0));
    //init
    const int length=1024;
    int a[length], b[length];
    for (int i=0;i<length;++i){
        a[i]=i;
        b[i]=i*i;
    }
    int *datas=NULL;
    CUDA_CALL(hipMalloc((void **)&datas, length * sizeof(int)));



    return 0;
}


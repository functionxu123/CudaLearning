#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <algorithm>
#include <cassert>

using namespace std;

// 用宏变长参数来实现
#define CUDA_CALL(...) {hipError_t _cuda_tep_set_not_repeat_a=(__VA_ARGS__);if (_cuda_tep_set_not_repeat_a!=hipSuccess){printf("\nCUDA ERROR: %s (err_num=%d)\n", hipGetErrorString(_cuda_tep_set_not_repeat_a), _cuda_tep_set_not_repeat_a); hipDeviceReset(); assert(0);} }
#define CUDA_LAST_ERROR() CUDA_CALL(hipGetLastError())

#define SHOW_MAT(a, m,n) \
{\
  cout<<"ShowMat: "<<#a<<endl;\
  for (int i=0;i<m;++i){\
    for (int j=0;j<n;++j){\
      cout<<a[i*n+j]<<", ";\
    }\
    cout<<endl;\
  }\
}

__global__ void matmult_v1(float *a, float *b, float *c, int m, int n, int k){//a-> m*k  b->k*n

  int idx = blockDim.x * blockIdx.x + threadIdx.x;
  int idy=blockIdx.y*blockDim.y+threadIdx.y;
  int index=gridDim.x*blockDim.x*idy+idx;
  if (idx>=n || idy>=m) return;
  
  c[index]=0;
  for (int i=0;i<k;++i){
    c[index]+=a[idy*k+i]*b[idx+i*n];
  }
}

int main(){
  int m=4;
  int n=5;
  int k=3;

  float *A=new float[m*k]{1,2,3,4,5,6,7,8,9,10,11};
  float *B=new float[k*n]{1,0,1,0,1,0,0,1,1,0,1,1,0};
  float *C=new float[m*n]{0};

  float *ga, *gb, *gc;
  // GPU端分配内存
  hipMalloc((void**)&ga, m*k*sizeof(float));
  hipMalloc((void**)&gb, k*n*sizeof(float));
  hipMalloc((void**)&gc, m*n*sizeof(float));

  // CPU的数据拷贝到GPU端
  hipMemcpy(ga, A, m*k*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(gb, B, k*n*sizeof(float), hipMemcpyHostToDevice);
  //cudaMemcpy(gc, C, size, cudaMemcpyHostToDevice);

  // 定义kernel执行配置，（1024*1024/512）个block，每个block里面有512个线程
  dim3 dimBlock(2,3);
  dim3 dimGrid(2,2);

  // 执行kernel
  matmult_v1<<<dimGrid, dimBlock>>>(ga, gb, gc, m,n,k);
  CUDA_LAST_ERROR();

  //cudaMemcpy ( void* dst, const void* src, size_t count, cudaMemcpyKind kind )
  hipMemcpy(C, gc, m*n*sizeof(float), hipMemcpyDeviceToHost);

  hipFree(ga);
  hipFree(gb);
  hipFree(gc);

  SHOW_MAT(A,m,k);
  SHOW_MAT(B,k,n);
  SHOW_MAT(C,m,n);

  delete []A;
  delete []B;
  delete []C;

  return 0;
}